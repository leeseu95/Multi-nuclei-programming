#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cmath>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

#include "../common/common.h"
#include <hip/hip_runtime.h>

using namespace std;

// input - input image one dimensional array
// ouput - output image one dimensional array
// width, height - width and height of the images
// colorWidthStep - number of color bytes (cols * colors)
// grayWidthStep - number of gray bytes 
__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep)
{
	// 2D Index of current thread
	const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
	const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

	// Only valid threads perform memory I/O
	if ((xIndex < width) && (yIndex < height))
	{
		//Location of colored pixel in input
		const int color_tid = yIndex * colorWidthStep + (3 * xIndex);

		//Location of gray pixel in output
		const int gray_tid = yIndex * grayWidthStep + xIndex;

		const unsigned char blue = input[color_tid];
		const unsigned char green = input[color_tid + 1];
		const unsigned char red = input[color_tid + 2];

		// The standard NTSC conversion formula that is used for calculating the effective luminance of a pixel (https://en.wikipedia.org/wiki/Grayscale#Luma_coding_in_video_systems)
		const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;
		
		// Alternatively, use an average
		//const float gray = (red + green + blue) / 3.f;

		output[gray_tid] = static_cast<unsigned char>(gray);
	}
}

void convert_to_gray(const cv::Mat& input, cv::Mat& output)
{
	cout << "Input image step: " << input.step << " rows: " << input.rows << " cols: " << input.cols << endl;
	// Calculate total number of bytes of input and output image
	// Step = cols * number of colors	
	size_t colorBytes = input.step * input.rows;
	size_t grayBytes = output.step * output.rows;

	unsigned char *d_input, *d_output;

	// Allocate device memory
	SAFE_CALL(hipMalloc<unsigned char>(&d_input, colorBytes), "CUDA Malloc Failed");
	SAFE_CALL(hipMalloc<unsigned char>(&d_output, grayBytes), "CUDA Malloc Failed");

	// Copy data from OpenCV input image to device memory
	SAFE_CALL(hipMemcpy(d_input, input.ptr(), colorBytes, hipMemcpyHostToDevice), "CUDA Memcpy Host To Device Failed");

	// Specify a reasonable block size
	const dim3 block(16, 16);

	// Calculate grid size to cover the whole image
	// const dim3 grid((input.cols + block.x - 1) / block.x, (input.rows + block.y - 1) / block.y);
	const dim3 grid((int)ceil((float)input.cols / block.x), (int)ceil((float)input.rows/ block.y));
	printf("bgr_to_gray_kernel<<<(%d, %d) , (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);
	
	// Launch the color conversion kernel
	bgr_to_gray_kernel <<<grid, block >>>(d_input, d_output, input.cols, input.rows, static_cast<int>(input.step), static_cast<int>(output.step));

	// Synchronize to check for any kernel launch errors
	SAFE_CALL(hipDeviceSynchronize(), "Kernel Launch Failed");

	// Copy back data from destination device meory to OpenCV output image
	SAFE_CALL(hipMemcpy(output.ptr(), d_output, grayBytes, hipMemcpyDeviceToHost), "CUDA Memcpy Host To Device Failed");

	// Free the device memory
	SAFE_CALL(hipFree(d_input), "CUDA Free Failed");
	SAFE_CALL(hipFree(d_output), "CUDA Free Failed");
}

int main(int argc, char *argv[])
{
	string imagePath;
	
	if(argc < 2)
		imagePath = "image.jpg";
  	else
  		imagePath = argv[1];

	// Read input image from the disk
	cv::Mat input = cv::imread(imagePath, CV_LOAD_IMAGE_COLOR);

	if (input.empty())
	{
		cout << "Image Not Found!" << std::endl;
		cin.get();
		return -1;
	}

	//Create output image
	cv::Mat output(input.rows, input.cols, CV_8UC1);

	//Call the wrapper function
	convert_to_gray(input, output);

	//Allow the windows to resize
	namedWindow("Input", cv::WINDOW_NORMAL);
	namedWindow("Output", cv::WINDOW_NORMAL);

	//Show the input and output
	imshow("Input", input);
	imshow("Output", output);

	//Wait for key press
	cv::waitKey();

	return 0;
}
